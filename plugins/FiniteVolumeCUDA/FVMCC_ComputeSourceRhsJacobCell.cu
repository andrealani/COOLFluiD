#include "hip/hip_runtime.h"
#include "FiniteVolumeCUDA/FVMCC_ComputeSourceRhsJacobCell.hh"
#include "Framework/MeshData.hh"
#include "Framework/BlockAccumulatorBaseCUDA.hh"
#include "Framework/CellConn.hh"
#include "Config/ConfigOptionPtr.hh"
#include "Framework/CudaDeviceManager.hh"
#include "Common/CUDA/CFVec.hh"
#include "Framework/CudaTimer.hh"
#include "FiniteVolume/CellData.hh"

#include "FiniteVolumeCUDA/FiniteVolumeCUDA.hh"
#include "Framework/MethodCommandProvider.hh"
#include "Framework/VarSetListT.hh"
#include "FiniteVolume/LaxFriedFlux.hh"
#include "FiniteVolume/LeastSquareP1PolyRec2D.hh"
#include "FiniteVolume/LeastSquareP1PolyRec3D.hh"
#include "FiniteVolume/BarthJesp.hh"
#include "FiniteVolume/Venktn2D.hh"

#include "MHD/MHD2DProjectionConsT.hh"
#include "MHD/MHD3DProjectionConsT.hh"
#include "MHD/MHD2DProjectionPrimT.hh"
#include "MHD/MHD3DProjectionPrimT.hh"
#include "MHD/MHDProjectionPrimToConsT.hh"
#include "FiniteVolumeMHD/LaxFriedFluxTanaka.hh"
#include "MHD/MHD2DProjectionVarSet.hh"
#include "MHD/MHD3DProjectionVarSet.hh"

#include "MultiFluidMHD/MultiFluidMHDVarSet.hh"
#include "MultiFluidMHD/EulerMFMHD2DHalfConsT.hh"
#include "MultiFluidMHD/EulerMFMHD2DHalfRhoiViTiT.hh"
#include "MultiFluidMHD/EulerMFMHD2DHalfRhoiViTiToConsT.hh"
#include "MultiFluidMHD/EulerMFMHD2DHalfConsToRhoiViTiT.hh"
#include "MultiFluidMHD/EulerMFMHD2DConsT.hh"
#include "MultiFluidMHD/EulerMFMHD2DRhoiViTiT.hh"
#include "MultiFluidMHD/EulerMFMHD2DRhoiViTiToConsT.hh"
#include "MultiFluidMHD/EulerMFMHD2DConsToRhoiViTiT.hh"
#include "FiniteVolumeMultiFluidMHD/AUSMPlusUpFluxMultiFluid.hh"
#include "FiniteVolumeMultiFluidMHD/AUSMFluxMultiFluid.hh"
#include "FiniteVolumeMultiFluidMHD/DriftWaves2DHalfTwoFluid.hh"
#include "FiniteVolumeMultiFluidMHD/HartmannSourceTerm.hh"

#include "Maxwell/Maxwell2DProjectionVarSet.hh"
#include "Maxwell/Maxwell2DProjectionConsT.hh"
#include "FiniteVolumeMaxwell/StegerWarmingMaxwellProjection2D.hh"
/*  // IA:  UNCOMMENT THIS FOR BUILDING THE SYSTEM ON THE GPU USING PARALUTION 1/2
#ifdef CF_HAVE_PARALUTION
#include "Paralution/ParalutionMatrix.hh"
#endif
*/
//////////////////////////////////////////////////////////////////////////////

using namespace COOLFluiD::Framework;
using namespace COOLFluiD::Common;
using namespace COOLFluiD::Config;
using namespace COOLFluiD::Physics::MHD;
using namespace COOLFluiD::Physics::Maxwell;
using namespace COOLFluiD::Physics::MultiFluidMHD;

//////////////////////////////////////////////////////////////////////////////

namespace COOLFluiD {

  namespace Numerics {

    namespace FiniteVolume {

//////////////////////////////////////////////////////////////////////////////



//Provider for AUSMPlusUpFlux with Source
#define FVMCC_MULTIFLUIDMHD_RHS_JACOB_PROV_AUSMPLUSUP_SOURCE(__dim__,__half__,__svars__,__uvars__,__sourceterm__,__limiter__,__nbBThreads__,__providerName__) \
MethodCommandProvider<FVMCC_ComputeSourceRhsJacobCell<AUSMPlusUpFluxMultiFluid<MultiFluidMHDVarSet<Maxwell##__dim__##ProjectionVarSet> >, \
			              VarSetListT<EulerMFMHD##__dim__##__half__##__svars__##T, EulerMFMHD##__dim__##__half__##__uvars__##T>, \
				      __sourceterm__<MultiFluidMHDVarSet<Maxwell##__dim__##ProjectionVarSet> >, \
				      LeastSquareP1PolyRec##__dim__ , __limiter__ , __nbBThreads__>, \
		      CellCenterFVMData, FiniteVolumeCUDAModule>	\
fvmcc_RhsJacobMultiFluidMHD##__limiter__##AUSMPlusUp##__dim__##__half__##__svars__##__uvars__##__sourceterm__##__nbBThreads__##Provider(__providerName__);

// 48 block threads (default)
//FVMCC_MULTIFLUIDMHD_RHS_JACOB_PROV_AUSMPLUSUP_SOURCE_BARTH(2D,Half,Cons,RhoiViTi,DriftWaves2DHalfTwoFluid,BarthJesp,48,"CellNumJacobBarthAUSMPlusUpEulerMFMHD2DHalfRhoiViTiDriftWavesTwoFluid")
FVMCC_MULTIFLUIDMHD_RHS_JACOB_PROV_AUSMPLUSUP_SOURCE(2D,Half,Cons,RhoiViTi,DriftWaves2DHalfTwoFluid,BarthJesp,48,"CellNumJacobBarthAUSMPlusUpEulerMFMHD2DHalfRhoiViTiDriftWavesTwoFluid")
FVMCC_MULTIFLUIDMHD_RHS_JACOB_PROV_AUSMPLUSUP_SOURCE(2D,Half,Cons,RhoiViTi,DriftWaves2DHalfTwoFluid,Venktn2D,48,"CellNumJacobVenktnAUSMPlusUpEulerMFMHD2DHalfRhoiViTiDriftWavesTwoFluid")

#undef FVMCC_MULTIFLUIDMHD_RHS_JACOB_PROV_AUSMPLUSUP_SOURCE


//////////////////////////////////////////////////////////////////////////////

template <typename PHYS>
HOST_DEVICE inline void setState(CFreal* state, CFreal* statePtr, 
				 CFreal* node, CFreal* nodePtr)
{
  // copy the state node data to shared memory
  for (CFuint i = 0; i < PHYS::DIM; ++i) {node[i] = nodePtr[i];}
  // copy the state data to shared memory
  for (CFuint i = 0; i < PHYS::NBEQS; ++i) {state[i] = statePtr[i];} 
}
      
//////////////////////////////////////////////////////////////////////////////
      
template <typename PHYS>
HOST_DEVICE inline void setFaceNormal(FluxData<PHYS>* fd, CFreal* normal)
{  
  CudaEnv::CFVecSlice<CFreal,PHYS::DIM> n(normal);
  const CFreal area = n.norm2();
  fd->setFaceArea(area);
  const CFreal ovArea = 1./area;
  CudaEnv::CFVecSlice<CFreal,PHYS::DIM> un(fd->getUnitNormal());
  for (CFuint i = 0; i < PHYS::DIM; ++i) {
    un[i] = n[i]*ovArea;
  }
}
      
//////////////////////////////////////////////////////////////////////////////

template <typename PHYS, typename PTR>
HOST_DEVICE void setFluxData(const CFuint f, const CFint stype, 
			     const CFuint stateID, const CFuint cellID, 
			     KernelData<CFreal>* kd, FluxData<PHYS>* fd,
			     PTR cellFaces)
{  
  fd->setStateID(RIGHT, stateID);
  CFreal* statePtrR = (stype > 0) ? &kd->states[stateID*PHYS::NBEQS] : &kd->ghostStates[stateID*PHYS::NBEQS];  
  CFreal* nodePtrR = (stype > 0) ? &kd->centerNodes[stateID*PHYS::DIM] : &kd->ghostNodes[stateID*PHYS::DIM];  
  setState<PHYS>(fd->getState(RIGHT), statePtrR, fd->getNode(RIGHT), nodePtrR);
  
  fd->setIsBFace(stype < 0);
  fd->setStateID(LEFT, cellID);
  const CFuint faceID = cellFaces[f*kd->nbCells + cellID];
  fd->setIsOutward(kd->isOutward[faceID] == cellID);
  
  CFreal* statePtrL = &kd->states[cellID*PHYS::NBEQS];
  CFreal* nodePtrL = &kd->centerNodes[cellID*PHYS::DIM];
  setState<PHYS>(fd->getState(LEFT), statePtrL, fd->getNode(LEFT), nodePtrL);
  setFaceNormal<PHYS>(fd, &kd->normals[faceID*PHYS::DIM]);
}

//////////////////////////////////////////////////////////////////////////////

template <typename T, CFuint SIZE>
void print(const std::string& name, T* array) 
{
  std::cout << name << " = \n";
  for (CFuint i = 0; i < SIZE; ++i) {
    std::cout.precision(10); std::cout << array[i] << " ";
  }
  std::cout << "\n";
}
      
//////////////////////////////////////////////////////////////////////////////

template <typename T, CFuint SIZE>
void printArray(T* array) 
{
  for (CFuint i = 0; i < SIZE; ++i) {
    std::cout << array[i] << " ";
  }
  std::cout << "\n";
}

//////////////////////////////////////////////////////////////////////////////

template <typename MODEL>
HOST_DEVICE void computeFaceCentroid(const CellData::Itr* cell, const CFuint faceIdx, 
				     const CFreal* nodes, CFreal* midFaceCoord)
{  
  CudaEnv::CFVecSlice<CFreal, MODEL::DIM> coord(midFaceCoord);
  coord = 0.;
  const CFuint nbFaceNodes = cell->getNbFaceNodes(faceIdx);
  const CFreal ovNbFaceNodes = 1./(static_cast<CFreal>(nbFaceNodes));
  for (CFuint n = 0; n < nbFaceNodes; ++n) {
    const CFuint cellNodeID = cell->getNodeID(faceIdx, n);
    const CFuint nodeID = cell->getNodeID(faceIdx,n);
    const CFreal* faceNode = &nodes[nodeID*MODEL::DIM];
    for (CFuint d = 0; d < MODEL::DIM; ++d) {
      coord[d] += faceNode[d];
    }
  }
  coord *= ovNbFaceNodes;
}

//////////////////////////////////////////////////////////////////////////////

template <typename PHYS, typename POLYREC>
__global__ void computeGradientsKernel(typename POLYREC::BASE::template DeviceConfigOptions<NOTYPE>* dcor,
				       const CFuint nbCells,
				       CFreal* states, 
				       CFreal* nodes,
				       CFreal* centerNodes,
				       CFreal* ghostStates,
				       CFreal* ghostNodes,
				       CFreal* uX,
				       CFreal* uY,
				       CFreal* uZ,
				       CFreal* limiter,
				       CFreal* updateCoeff, 
				       CFreal* rhs,
				       CFreal* normals,
				       CFint* isOutward,
				       const CFuint* cellInfo,
				       const CFuint* cellStencil,
				       const CFuint* cellFaces,
				       const CFuint* cellNodes,
				       const CFint*  neighborTypes,
				       const Framework::CellConn* cellConn)
{    
  // each thread takes care of computing the gradient for one single cell
  const int cellID = threadIdx.x + blockIdx.x*blockDim.x;
  
  if (cellID < nbCells) {    
    KernelData<CFreal> kd (nbCells, states, nodes, centerNodes, ghostStates, ghostNodes, updateCoeff, 
			   rhs, normals, uX, uY, uZ, isOutward);
    
    // compute and store cell gradients at once 
    POLYREC polyRec(dcor);
    CellData cells(nbCells, cellInfo, cellStencil, cellFaces, cellNodes, neighborTypes, cellConn);
    CellData::Itr cell = cells.getItr(cellID);
    polyRec.computeGradients(&states[cellID*PHYS::NBEQS], &centerNodes[cellID*PHYS::DIM], &kd, &cell);
  }
}
      
//////////////////////////////////////////////////////////////////////////////

template <typename PHYS, typename POLYREC, typename LIMITER>
__global__ void computeLimiterKernel(typename LIMITER::BASE::template DeviceConfigOptions<NOTYPE>* dcol,
				     typename POLYREC::BASE::template DeviceConfigOptions<NOTYPE>* dcor,
				     const CFuint nbCells,
				     CFreal* states, 
				     CFreal* nodes,
				     CFreal* centerNodes,
				     CFreal* ghostStates,
				     CFreal* ghostNodes,
				     CFreal* uX,
				     CFreal* uY,
				     CFreal* uZ,
				     CFreal* limiter,
				     CFreal* updateCoeff, 
				     CFreal* rhs,
				     CFreal* normals,
				     CFint* isOutward,
				     const CFuint* cellInfo,
				     const CFuint* cellStencil,
				     const CFuint* cellFaces,
				     const CFuint* cellNodes,
				     const CFint*  neighborTypes,
				     const Framework::CellConn* cellConn)
{    
  // each thread takes care of computing the gradient for one single cell
  const int cellID = threadIdx.x + blockIdx.x*blockDim.x;
 
  if (cellID < nbCells) {    
    // compute all cell quadrature points at once (size of this array is overestimated)
    CFreal midFaceCoord[PHYS::DIM*PHYS::DIM*2];
    
    CellData cells(nbCells, cellInfo, cellStencil, cellFaces, cellNodes, neighborTypes, cellConn);
    CellData::Itr cell = cells.getItr(cellID);
    const CFuint nbFacesInCell = cell.getNbFacesInCell();
    for (CFuint f = 0; f < nbFacesInCell; ++f) { 
      computeFaceCentroid<PHYS>(&cell, f, nodes, &midFaceCoord[f*PHYS::DIM]);
    }
    
    // compute cell-based limiter at once
    KernelData<CFreal> kd (nbCells, states, nodes, centerNodes, ghostStates, ghostNodes, updateCoeff, 
			   rhs, normals, uX, uY, uZ, isOutward);
    LIMITER limt(dcol);
    
    if (dcor->currRes > dcor->limitRes && (dcor->limitIter > 0 && dcor->currIter < dcor->limitIter)) {	
      limt.limit(&kd, &cell, &midFaceCoord[0], &limiter[cellID*PHYS::NBEQS]);
    }
    else {
      if (!dcor->freezeLimiter) {
	// historical modification of the limiter
	CudaEnv::CFVec<CFreal,PHYS::NBEQS> tmpLimiter;
	limt.limit(&kd, &cell, &midFaceCoord[0], &tmpLimiter[0]);
	CFuint currID = cellID*PHYS::NBEQS;
	for (CFuint iVar = 0; iVar < PHYS::NBEQS; ++iVar, ++currID) {
	  limiter[currID] = min(tmpLimiter[iVar],limiter[currID]);
	}
      }
    }
  }
}

template <typename SCHEME, typename POLYREC, typename LIMITER, typename SOURCE>
__global__ void computeFluxSourceJacobianKernel(typename SCHEME::BASE::template DeviceConfigOptions<NOTYPE>* dcof,
					  typename POLYREC::BASE::template DeviceConfigOptions<NOTYPE>* dcor,
					  typename LIMITER::BASE::template DeviceConfigOptions<NOTYPE>* dcol,
					  typename NumericalJacobian::DeviceConfigOptions<typename SCHEME::MODEL>* dcon,
					  typename SCHEME::MODEL::PTERM::template DeviceConfigOptions<NOTYPE>* dcop,
                                          typename SOURCE::BASE::template DeviceConfigOptions<NOTYPE>* dcos,
					  CFreal* volume,
					  const CFuint nbCells,
					  const CFuint startCellID,
					  CFreal* states, 
					  CFreal* nodes,
					  CFreal* centerNodes,
					  CFreal* ghostStates,
					  CFreal* ghostNodes,
					  CFreal* blockJacob,
					  CFuint* blockStart,
					  CFreal* uX,
					  CFreal* uY,
					  CFreal* uZ,
					  CFreal* limiter,
					  CFreal* updateCoeff, 
					  CFreal* rhs,
					  CFreal* normals,
					  CFint* isOutward,
					  const CFuint* cellInfo,
					  const CFuint* cellStencil,
					  const CFuint* cellFaces,
					  const CFuint* cellNodes,
					  const CFint* neighborTypes,
					  const Framework::CellConn* cellConn,
			           	  CFreal ResFactor, bool IsAxisymmetric)
{  
  typedef typename SCHEME::MODEL PHYS;

  // each thread takes care of computing the gradient for one single cell
  const int cellID = threadIdx.x + blockIdx.x*blockDim.x + startCellID;
  if (cellID < nbCells) {
    // compute and store cell gradients at once 
    POLYREC polyRec(dcor);
    SCHEME  fluxScheme(dcof);
    LIMITER limt(dcol);
    SOURCE Source(dcos);
    NumericalJacobian::DeviceFunc<PHYS> numJacob(dcon);
    KernelData<CFreal> kd (nbCells, states, nodes, centerNodes, ghostStates, 
			   ghostNodes, updateCoeff, rhs, normals, uX, uY, uZ, isOutward);
    
    // compute all cell quadrature points at once (array size can be overestimated in 3D)
    const CFuint MAX_NB_FACES = PHYS::DIM*2;
    CFreal midFaceCoord[PHYS::DIM*MAX_NB_FACES];
    CudaEnv::CFVec<CFreal,PHYS::NBEQS> fluxDiff;
    CudaEnv::CFVec<CFreal,PHYS::NBEQS> resBkp;
    //FluxData<PHYS> currFd; currFd.initialize();
    FluxData<PHYS> fd; fd.initialize();
    FluxData<PHYS>* currFd = &fd;
    typename SCHEME::MODEL pmodel(dcop);
    
    // reset the rhs and update coefficients to 0
    CudaEnv::CFVecSlice<CFreal,PHYS::NBEQS> res(&rhs[cellID*PHYS::NBEQS]);
    res = 0.;
    updateCoeff[cellID] = 0.;
   

    CFreal invR = 1.0;
    if (IsAxisymmetric) {     
      printf("IsAxisymmetric=true not implemented \n");
      //invR /= abs(currCell->getState(0)->getCoordinates()[YY]); //It just need the y-component (easy addition)
    }

    CFreal factor = invR*volume[cellID]*ResFactor;

    //Arrays needed for the source jacobian
    CudaEnv::CFVec<CFreal,SOURCE::MODEL::NBEQS> SourceDiff;
    CudaEnv::CFVec<CFreal,SOURCE::MODEL::NBEQS> sourceBkp;
    CudaEnv::CFVec<CFreal,SOURCE::MODEL::NBEQS> source;
    source = 0.0;
    sourceBkp = 0.0;    

    CellData cells(nbCells, cellInfo, cellStencil, cellFaces, cellNodes, neighborTypes, cellConn);
    CellData::Itr cell = cells.getItr(cellID);
    const CFuint nbFacesInCell = cell.getNbActiveFacesInCell();
    const CFuint nbRows = nbFacesInCell + 1;
    const CFuint bStartCellID = blockStart[cellID];

    // this block accumulator represents a column block (nbFaces+1 x 1)
    BlockAccumulatorBaseCUDA acc(nbRows, 1, PHYS::NBEQS, &blockJacob[bStartCellID]);
    acc.reset();
    
    // compute the face flux and flux numerical jacobian within the same loop
    for (CFuint f = 0; f < nbFacesInCell; ++f) { 
      const CFint stype = cell.getNeighborType(f);
      if (stype != 0) { // skip all partition and boundary faces
	const CFuint stateID = cell.getNeighborID(f);
	setFluxData(f, stype, stateID, cellID, &kd, currFd, cellFaces);
	// compute face quadrature points (face centroids)
	CFreal* faceCenters = &midFaceCoord[f*PHYS::DIM];
	computeFaceCentroid<PHYS>(&cell, f, nodes, faceCenters);
	
	// extrapolate solution on quadrature points on both sides of the face
	polyRec.extrapolateOnFace(currFd, faceCenters, uX, uY, uZ, limiter);
        fluxScheme.prepareComputation(currFd, &pmodel);
	fluxScheme(currFd, &pmodel); // compute the convective flux across the face
	
	for (CFuint iEq = 0; iEq < PHYS::NBEQS; ++iEq) {
	  const CFreal value = currFd->getResidual()[iEq];
	  res[iEq]   -= value;  // update the residual 
	  resBkp[iEq] = value;  // backup the current face-based residual
	}
	
	// update the update coefficient
	updateCoeff[cellID] += currFd->getUpdateCoeff();
		
	// only contribution from internal faces is computed here  
	if (stype > 0) { 	  
	  currFd->setIsPerturb(true);
	  // flux jacobian computation
	  for (CFuint iVar = 0; iVar < PHYS::NBEQS; ++iVar) {
	    const CFreal rstateBkpL = currFd->getRstate(LEFT)[iVar];
            //const CFreal stateBkp = currFd.getState(LEFT)[iVar];
	    // here we perturb the current variable for the left cell state
	    numJacob.perturb(iVar, &currFd->getState(LEFT)[iVar]);
	    
	    // extrapolate solution on quadrature points on both sides of the face
	    polyRec.extrapolateOnFace(iVar, currFd, faceCenters, uX, uY, uZ, limiter);

	    fluxScheme(currFd, &pmodel); // compute the convective flux across the face
            fluxDiff = 0.;    
	    // compute the numerical jacobian of the flux
	    CudaEnv::CFVecSlice<CFreal,PHYS::NBEQS> resPert(currFd->getResidual());
	    numJacob.computeDerivative(&resBkp, &resPert, &fluxDiff);


	    // contribution to the row corresponding of the current cell
	    // this subblock gets all contributions from all face cells
	    acc.addValues(0, 0, iVar, &fluxDiff[0]);
	    
	    // contribution to row corresponding to the f+1 cell: 
	    // this is the flux jacobian contribution for the neighbor cells
	    // due to the currently perturbed cell state and is opposite in sign
	    // because the outward normal for neighbors is inward for the current cell
	    fluxDiff *= -1.0;
	    acc.addValues(f+1, 0, iVar, &fluxDiff[0]);
	    
	    // restore perturbed states
	    currFd->getRstate(LEFT)[iVar] = rstateBkpL;
  	    numJacob.restore(&currFd->getState(LEFT)[iVar]);
	  }
	  
	  currFd->setIsPerturb(false);
	}
      }
    }
   
    //Source computation
    CudaEnv::CFVecSlice<CFreal,SOURCE::MODEL::NBEQS> state(&states[cellID*SOURCE::MODEL::NBEQS]);
    Source(&state[0], &pmodel, &source[0]);   //Source term computation
    source *= factor;    

    for (CFuint iEq = 0; iEq < SOURCE::MODEL::NBEQS; ++iEq) {  //Add source term to the RHS and create backup for the derivatives
       const CFreal value = source[iEq]; 
       res[iEq] += value;  
       sourceBkp[iEq] = value;
    }


    //Source Jacobian computation
    for (CFuint iVar = 0; iVar < SOURCE::MODEL::NBEQS; ++iVar) {
      const CFreal stateBkp = state[iVar];
      // here we perturb the current variable for the state
      numJacob.perturb(iVar, &state[iVar]);
	    
      //Computation of the source with the perturbed state
      Source(&state[0], &pmodel, &source[0]);
	    
      //Compute the numerical derivative
      source *= factor;
      SourceDiff = 0.0; 
      numJacob.computeDerivative(&sourceBkp, &source, &SourceDiff);
      SourceDiff *= -1.; 

      acc.addValues(0, 0, iVar, &SourceDiff[0]);   //Add values to the block accumulator

      // restore perturbed states
      state[iVar] = stateBkp;
//	    numJacob.restore(&state[iVar]); // IA: Why does this not work? Anyway, the previous line is an alternative that works..
    }


  }
}










template <typename SCHEME, typename POLYREC, typename LIMITER, typename SOURCE>
__global__ void computeFluxSourceJacobianKernelParalution(typename SCHEME::BASE::template DeviceConfigOptions<NOTYPE>* dcof,
					  typename POLYREC::BASE::template DeviceConfigOptions<NOTYPE>* dcor,
					  typename LIMITER::BASE::template DeviceConfigOptions<NOTYPE>* dcol,
					  typename NumericalJacobian::DeviceConfigOptions<typename SCHEME::MODEL>* dcon,
					  typename SCHEME::MODEL::PTERM::template DeviceConfigOptions<NOTYPE>* dcop,
                                          typename SOURCE::BASE::template DeviceConfigOptions<NOTYPE>* dcos,
					  CFreal* volume,
					  const CFuint nbCells,
					  const CFuint startCellID,
					  CFreal* states, 
					  CFreal* nodes,
					  CFreal* centerNodes,
					  CFreal* ghostStates,
					  CFreal* ghostNodes,
//					  CFreal* blockJacob,
					  CFuint* blockStart,
   					  CFint* rowoff,
   					  CFint* col,
   					  CFreal* val,
					  CFreal* uX,
					  CFreal* uY,
					  CFreal* uZ,
					  CFreal* limiter,
					  CFreal* updateCoeff, 
					  CFreal* rhs,
					  CFreal* normals,
					  CFint* isOutward,
					  const CFuint* cellInfo,
					  const CFuint* cellStencil,
					  const CFuint* cellFaces,
					  const CFuint* cellNodes,
					  const CFint* neighborTypes,
					  const Framework::CellConn* cellConn,
			           	  CFreal ResFactor, bool IsAxisymmetric)
{  
  typedef typename SCHEME::MODEL PHYS;

  // each thread takes care of computing the gradient for one single cell
  const int cellID = threadIdx.x + blockIdx.x*blockDim.x + startCellID;
  if (cellID < nbCells) {
 
    // compute and store cell gradients at once 
    POLYREC polyRec(dcor);
    SCHEME  fluxScheme(dcof);
    LIMITER limt(dcol);
    SOURCE Source(dcos);
    NumericalJacobian::DeviceFunc<PHYS> numJacob(dcon);


    KernelData<CFreal> kd (nbCells, states, nodes, centerNodes, ghostStates, 
			   ghostNodes, updateCoeff, rhs, normals, uX, uY, uZ, isOutward);
    
    // compute all cell quadrature points at once (array size can be overestimated in 3D)
    const CFuint MAX_NB_FACES = PHYS::DIM*2;
    CFreal midFaceCoord[PHYS::DIM*MAX_NB_FACES];
    CudaEnv::CFVec<CFreal,PHYS::NBEQS> fluxDiff;
    CudaEnv::CFVec<CFreal,PHYS::NBEQS> resBkp;
    FluxData<PHYS> currFd; currFd.initialize();
    typename SCHEME::MODEL pmodel(dcop);
    
    // reset the rhs and update coefficients to 0
    CudaEnv::CFVecSlice<CFreal,PHYS::NBEQS> res(&rhs[cellID*PHYS::NBEQS]);
    res = 0.;
    updateCoeff[cellID] = 0.;
   

    CFreal invR = 1.0;
    if (IsAxisymmetric) {     
      printf("IsAxisymmetric=true not implemented \n");
      //invR /= abs(currCell->getState(0)->getCoordinates()[YY]); //It just need the y-component (easy addition)
    }

    CFreal factor = invR*volume[cellID]*ResFactor;

    //Arrays needed for the source jacobian
    CudaEnv::CFVec<CFreal,SOURCE::MODEL::NBEQS> SourceDiff;
    CudaEnv::CFVec<CFreal,SOURCE::MODEL::NBEQS> sourceBkp;
    CudaEnv::CFVec<CFreal,SOURCE::MODEL::NBEQS> source;
    source = 0.0;
    sourceBkp = 0.0;    

    CellData cells(nbCells, cellInfo, cellStencil, cellFaces, cellNodes, neighborTypes, cellConn);
    CellData::Itr cell = cells.getItr(cellID);
    const CFuint nbFacesInCell = cell.getNbActiveFacesInCell();
    
  
    //Compute the index for the diagonal blocks
    const CFint nb = PHYS::NBEQS;

    CFuint RowPositionDiag = rowoff[cellID*nb];           //In this case we are looking for the diagonal block
    CFuint RowPositionPlusOneDiag = rowoff[cellID*nb + 1];
    CFuint mmDiag = (RowPositionPlusOneDiag-RowPositionDiag)/nb;
    CFuint IndexCSRDiag = RowPositionDiag;

  
    // Number of valid faces
    CFint Nf = 1;

    // compute the face flux and flux numerical jacobian within the same loop
    for (CFuint f = 0; f < nbFacesInCell; ++f) { 
      const CFint stype = cell.getNeighborType(f);
      
      if (stype != 0) { // skip all partition and boundary faces
	const CFuint stateID = cell.getNeighborID(f);
	setFluxData(f, stype, stateID, cellID, &kd, &currFd, cellFaces);
	
	// compute face quadrature points (face centroids)
	CFreal* faceCenters = &midFaceCoord[f*PHYS::DIM];
	computeFaceCentroid<PHYS>(&cell, f, nodes, faceCenters);
	
	// extrapolate solution on quadrature points on both sides of the face
	polyRec.extrapolateOnFace(&currFd, faceCenters, uX, uY, uZ, limiter);
        fluxScheme.prepareComputation(&currFd, &pmodel);
	fluxScheme(&currFd, &pmodel); // compute the convective flux across the face
	
	for (CFuint iEq = 0; iEq < PHYS::NBEQS; ++iEq) {
	  const CFreal value = currFd.getResidual()[iEq];
	  res[iEq]   -= value;  // update the residual 
	  resBkp[iEq] = value;  // backup the current face-based residual
	}
	
	// update the update coefficient
	updateCoeff[cellID] += currFd.getUpdateCoeff();
		
	// only contribution from internal faces is computed here  
	if (stype > 0) { 	  
	  currFd.setIsPerturb(true);
	  // flux jacobian computation
	  for (CFuint iVar = 0; iVar < PHYS::NBEQS; ++iVar) {
	    // here we perturb the current variable for the left cell state
	    numJacob.perturb(iVar, &currFd.getState(LEFT)[iVar]);
	    
	    // extrapolate solution on quadrature points on both sides of the face
	    const CFreal rstateBkpL = currFd.getRstate(LEFT)[iVar];
	    polyRec.extrapolateOnFace(iVar, &currFd, faceCenters, uX, uY, uZ, limiter);
	    fluxScheme(&currFd, &pmodel); // compute the convective flux across the face
	    
	    // compute the numerical jacobian of the flux
	    CudaEnv::CFVecSlice<CFreal,PHYS::NBEQS> resPert(currFd.getResidual());
	    numJacob.computeDerivative(&resBkp, &resPert, &fluxDiff);
	    
	    // contribution to the row corresponding of the current cell
	    // this subblock gets all contributions from all face cells
	    //acc.addValues(0, 0, iVar, &fluxDiff[0]);

            // IA: this is equivalent to: acc.addValues(0, 0, iVar, &fluxDiff[0]);

            for (CFint nbi=0; nbi<PHYS::NBEQS; nbi++){
               col[IndexCSRDiag+nbi*nb*mmDiag+iVar] = cellID*nb + iVar; 
               val[IndexCSRDiag+nbi*nb*mmDiag+iVar] += fluxDiff[nbi];
            }
	    
	    // contribution to row corresponding to the f+1 cell: 
	    // this is the flux jacobian contribution for the neighbor cells
	    // due to the currently perturbed cell state and is opposite in sign
	    // because the outward normal for neighbors is inward for the current cell
	    //fluxDiff *= -1.0;
//	    acc.addValues(f+1, 0, iVar, &fluxDiff[0]); 
	    
	    // restore perturbed states
	    currFd.getRstate(LEFT)[iVar] = rstateBkpL;
	    numJacob.restore(&currFd.getState(LEFT)[iVar]);

            // IA: this is equivalent to: acc.addValues(f+1, 0, iVar, &fluxDiff[0]);
            CFuint RowPosition = rowoff[stateID*nb]; //stateID is the ID of the neighbour
            CFuint RowPositionPlusOne = rowoff[stateID*nb + 1];
            CFuint mm = (RowPositionPlusOne-RowPosition)/nb;
            CFint IndexCSR = -1;


            for (CFuint mii=0; mii<mm; mii++){
               if(col[RowPosition+mii*nb] == cellID*nb || col[RowPosition+mii*nb] == -1){
                  IndexCSR = RowPosition+mii*nb;
               }
            }
            for (CFint nbi=0; nbi<PHYS::NBEQS; nbi++){
               col[IndexCSR+nbi*nb*mm+iVar] = cellID*nb + iVar;
               val[IndexCSR+nbi*nb*mm+iVar] -= fluxDiff[nbi];  //fluxDiff *= -1.0;
            } 

	  }
	  Nf++;
	  currFd.setIsPerturb(false);
	}
      }
    }
   

    //Source computation
    CudaEnv::CFVecSlice<CFreal,SOURCE::MODEL::NBEQS> state(&states[cellID*SOURCE::MODEL::NBEQS]);
    Source(&state[0], &pmodel, &source[0]);   //Source term computation
    source *= factor;    

    for (CFuint iEq = 0; iEq < SOURCE::MODEL::NBEQS; ++iEq) {  //Add source term to the RHS and create backup for the derivatives
       const CFreal value = source[iEq]; 
       res[iEq] += value;  
       sourceBkp[iEq] = value;
    }


    //Source Jacobian computation
    for (CFuint iVar = 0; iVar < SOURCE::MODEL::NBEQS; ++iVar) {
      const CFreal stateBkp = state[iVar];
      // here we perturb the current variable for the state
      numJacob.perturb(iVar, &state[iVar]);
	    
      //Computation of the source with the perturbed state
      Source(&state[0], &pmodel, &source[0]);
	    
      //Compute the numerical derivative
      source *= factor;
      SourceDiff = 0.0; 
      numJacob.computeDerivative(&sourceBkp, &source, &SourceDiff);
      SourceDiff *= -1.; 

//      acc.addValues(0, 0, iVar, &SourceDiff[0]);   //Add values to the block accumulator
      for (CFint nbi=0; nbi<PHYS::NBEQS; nbi++){
         col[IndexCSRDiag+nbi*nb*mmDiag+iVar] = cellID*nb + iVar; 
         val[IndexCSRDiag+nbi*nb*mmDiag+iVar] += SourceDiff[nbi];
      }



      // restore perturbed states
      state[iVar] = stateBkp;
    }


  }

}





















  
template <typename SCHEME, typename SOURCE, typename POLYREC, typename LIMITER>
void computeFluxSourceJacobianCPU(typename SCHEME::BASE::template DeviceConfigOptions<NOTYPE>* dcof,
			    typename POLYREC::BASE::template DeviceConfigOptions<NOTYPE>* dcor,
			    typename LIMITER::BASE::template DeviceConfigOptions<NOTYPE>* dcol,
			    typename NumericalJacobian::DeviceConfigOptions<typename SCHEME::MODEL>* dcon,
			    typename SCHEME::MODEL::PTERM::template DeviceConfigOptions<NOTYPE>* dcop,
                            typename SOURCE::BASE::template DeviceConfigOptions<NOTYPE>* dcos,
			    const CFuint nbCells,
			    CFreal* states, 
                            CFreal* volumes,
			    CFreal* nodes,
			    CFreal* centerNodes,
			    CFreal* ghostStates,
			    CFreal* ghostNodes, 
			    CFreal* blockJacob,
			    CFuint* blockStart,
			    CFreal* uX,
			    CFreal* uY,
			    CFreal* uZ,
			    CFreal* limiter,
			    CFreal* updateCoeff, 
			    CFreal* rhs,
			    CFreal* normals,
			    CFint* isOutward,
			    const CFuint* cellInfo,
			    const CFuint* cellStencil,
			    const CFuint* cellFaces,
			    const CFuint* cellNodes,
			    const CFint* neighborTypes,
			    const Framework::CellConn* cellConn,
                            CFreal ResFactor, bool IsAxisymmetric)
{ 
  using namespace std;
  
  typedef typename SCHEME::MODEL PHYS;
  
  CudaEnv::CudaTimer& timer = CudaEnv::CudaTimer::getInstance();
  timer.start();
  
  FluxData<PHYS> fd; fd.initialize();
  FluxData<PHYS>* currFd = &fd;
  cf_assert(currFd != CFNULL);
  SCHEME fluxScheme(dcof);
  POLYREC polyRec(dcor);
  LIMITER limt(dcol);
  
  CellData cells(nbCells, cellInfo, cellStencil, cellFaces, cellNodes, neighborTypes, cellConn);
  KernelData<CFreal> kd(nbCells, states, nodes, centerNodes, ghostStates, ghostNodes, updateCoeff, 
			rhs, normals, uX, uY, uZ, isOutward);
  
  const CFuint MAX_NB_FACES = PHYS::DIM*2;
  CFreal midFaceCoord[PHYS::DIM*MAX_NB_FACES];
  CudaEnv::CFVec<CFreal,PHYS::NBEQS> fluxDiff;
  CudaEnv::CFVec<CFreal,PHYS::NBEQS> resBkp;
  CudaEnv::CFVec<CFreal,PHYS::NBEQS> tmpLimiter;
  NumericalJacobian::DeviceFunc<PHYS> numJacob(dcon);
  PHYS pmodel(dcop);
  

  CudaEnv::CFVec<CFreal,PHYS::NBEQS> source;
  CudaEnv::CFVec<CFreal,SOURCE::MODEL::NBEQS> SourceDiff;
  CudaEnv::CFVec<CFreal,SOURCE::MODEL::NBEQS> sourceBkp;
  SOURCE Source(dcos);



  // compute the cell-based gradients
  for (CFuint cellID = 0; cellID < nbCells; ++cellID) {
    CellData::Itr cell = cells.getItr(cellID);
    polyRec.computeGradients(&states[cellID*PHYS::NBEQS], &centerNodes[cellID*PHYS::DIM], &kd, &cell);
  }
  
  // printGradients<PHYS::NBEQS>(uX, uY, uZ, nbCells);
  CFLog(VERBOSE, "FVMCC_ComputeSourceRhsJacobCell::computeFluxJacobianCPU() => computing gradients took " << timer.elapsed() << " s\n");
  timer.start();
  
  // compute the cell based limiter
  for (CellData::Itr cell = cells.begin(); cell <= cells.end(); ++cell) {
    // compute all cell quadrature points at once (size of this array is overestimated)
    const CFuint nbFacesInCell = cell.getNbFacesInCell();
    for (CFuint f = 0; f < nbFacesInCell; ++f) { 
      computeFaceCentroid<PHYS>(&cell, f, nodes, &midFaceCoord[f*PHYS::DIM]);
    }
    
    const CFuint cellID = cell.getCellID();
    if (dcor->currRes > dcor->limitRes && (dcor->limitIter > 0 && dcor->currIter < dcor->limitIter)) {	
      // compute cell-based limiter
      limt.limit(&kd, &cell, &midFaceCoord[0], &limiter[cellID*PHYS::NBEQS]);
    }
    else {
      if (!dcor->freezeLimiter) {
	// historical modification of the limiter
	limt.limit(&kd, &cell, &midFaceCoord[0], &tmpLimiter[0]);
	CFuint currID = cellID*PHYS::NBEQS;
	for (CFuint iVar = 0; iVar < PHYS::NBEQS; ++iVar, ++currID) {
	  limiter[currID] = min(tmpLimiter[iVar],limiter[currID]);
	}
      }
    }
  }
  
  // printLimiter<PHYS::NBEQS>(limiter, nbCells);
  
  CFLog(VERBOSE, "FVMCC_ComputeSourceRhsJacobCell::computeFluxJacobianCPU() => computing limiter took " << timer.elapsed() << " s\n");
  timer.start();
  
  // compute the fluxes and the jacobian
  for (CellData::Itr cell = cells.begin(); cell <= cells.end(); ++cell) {
    // reset the rhs and update coefficients to 0
    const CFuint cellID = cell.getCellID();
    CudaEnv::CFVecSlice<CFreal,PHYS::NBEQS> res(&rhs[cellID*PHYS::NBEQS]);
    res = 0.;
    updateCoeff[cellID] = 0.;
    
    const CFuint nbFacesInCell = cell.getNbActiveFacesInCell();
    const CFuint nbRows = nbFacesInCell + 1;
    const CFuint bStartCellID = blockStart[cellID];
        
    // this block accumulator represents a column block (nbFaces+1 x 1)
    BlockAccumulatorBaseCUDA acc(nbRows, 1, PHYS::NBEQS, &blockJacob[bStartCellID]);
    acc.reset();
    
    



    for (CFuint f = 0; f < nbFacesInCell; ++f) { 
      const CFint stype = cell.getNeighborType(f);
      if (stype != 0) { // skip all partition and boundary faces
	const CFuint stateID =  cell.getNeighborID(f);
	setFluxData(f, stype, stateID, cellID, &kd, currFd, cellFaces);

	// compute face quadrature points (centroid)
	CFreal* faceCenters = &midFaceCoord[f*PHYS::DIM];
	computeFaceCentroid<PHYS>(&cell, f, nodes, faceCenters);
	
	// extrapolate solution on quadrature points on both sides of the face
	polyRec.extrapolateOnFace(currFd, faceCenters, uX, uY, uZ, limiter);
	
        fluxScheme.prepareComputation(currFd, &pmodel);
	fluxScheme(currFd, &pmodel); // compute the convective flux across the face

	for (CFuint iEq = 0; iEq < PHYS::NBEQS; ++iEq) {
	  const CFreal value = currFd->getResidual()[iEq];
	  res[iEq]   -= value;  // update the residual 
	  resBkp[iEq] = value;  // backup the current face-based residual
	}
	
	// update the update coefficient
	updateCoeff[cellID] += currFd->getUpdateCoeff();
	
        /////////////////////////////////////////////////
 



	// only contribution from internal faces is computed here  
	if (stype > 0) { 
	  currFd->setIsPerturb(true);
	  
	  // flux jacobian computation
	  for (CFuint iVar = 0; iVar < PHYS::NBEQS; ++iVar) {
	    // here we perturb the current variable for the left cell state
	    numJacob.perturb(iVar, &currFd->getState(LEFT)[iVar]);
	    
	    // extrapolate solution on quadrature points on both sides of the face
	    const CFreal rstateBkpL = currFd->getRstate(LEFT)[iVar];
	    polyRec.extrapolateOnFace(iVar, currFd, faceCenters, uX, uY, uZ, limiter);
	    fluxScheme(currFd, &pmodel); // compute the convective flux across the face
	    
	    // compute the numerical jacobian of the flux
	    CudaEnv::CFVecSlice<CFreal,PHYS::NBEQS> resPert(currFd->getResidual());
	    numJacob.computeDerivative(&resBkp, &resPert, &fluxDiff);


	    // flux is computed with the outward normal, so the sign is correct here
	    // contribution to the row corresponding of the current cell
	    // this subblock gets all contributions from all face cells
	    acc.addValues(0, 0, iVar, &fluxDiff[0]);
	    
	    // contribution to row corresponding to the f+1 cell: 
	    // this is the flux jacobian contribution for the neighbor cells
	    // due to the currently perturbed cell state and is opposite in sign
	    // because the outward normal for neighbors is inward for the current cell
	    fluxDiff *= -1.0;

	    acc.addValues(f+1, 0, iVar, &fluxDiff[0]);   
	    
	    // restore perturbed states
	    currFd->getRstate(LEFT)[iVar] = rstateBkpL;
	    numJacob.restore(&currFd->getState(LEFT)[iVar]);
	  }
	    
	  currFd->setIsPerturb(false);
	}
      }
    }

    //Source computation
    source = 0.;
    
    CudaEnv::CFVec<CFreal,SOURCE::MODEL::NBEQS> stateBkp;
    CudaEnv::CFVecSlice<CFreal,PHYS::NBEQS> state(&states[cellID*PHYS::NBEQS]);
    Source(&state[0], &pmodel, &source[0]);

    CFreal invR = 1.0;
    if (IsAxisymmetric) {   
      printf("IsAxissymmetric=true not implemeted \n");  
      //invR /= abs(currCell->getState(0)->getCoordinates()[YY]);  
    }
    CFreal factor = invR*volumes[cellID]*ResFactor;     

    source *= factor;
    for (CFuint iEq = 0; iEq < PHYS::NBEQS; ++iEq) { 
       res[iEq] += source[iEq];   
       sourceBkp[iEq] = source[iEq];
       stateBkp[iEq] = state[iEq];
    }
    
    /////////////////////////////////////////////

    //Source Jacobian computation
    for (CFuint iVar = 0; iVar < SOURCE::MODEL::NBEQS; ++iVar) {
      // here we perturb the current variable for the state
      numJacob.perturb(iVar, &state[iVar]);
	    
      //Computation of the source with the perturbed state
      Source(&state[0], &pmodel, &source[0]);
	    
      //Compute the numerical derivative
      source *= factor;
      SourceDiff = 0.0; 
      numJacob.computeDerivative(&sourceBkp, &source, &SourceDiff);
	    
      // contribution to the row corresponding of the current cell
      // this subblock gets all contributions from all face cells
       
      SourceDiff *= -1.;

      acc.addValues(0, 0, iVar, &SourceDiff[0]);   

      // restore perturbed states
      state[iVar] = stateBkp[iVar];
      numJacob.restore(&state[iVar]);
    }

  //////////////////////////////


  } 
  
  CFLog(VERBOSE, "FVMCC_ComputeSourceRhsJacobCell::computeFluxJacobianCPU()  took " << timer.elapsed() << " s\n");
}

//////////////////////////////////////////////////////////////////////////////

template <typename SCHEME, typename PHYSICS, typename SOURCE, typename POLYREC, typename LIMITER, CFuint NB_BLOCK_THREADS>
void FVMCC_ComputeSourceRhsJacobCell<SCHEME,PHYSICS,SOURCE,POLYREC,LIMITER,NB_BLOCK_THREADS>::execute()
{
  using namespace std;
  using namespace COOLFluiD::Framework;
  using namespace COOLFluiD::Common;
  
  CFTRACEBEGIN;
  
  CFLog(VERBOSE, "FVMCC_ComputeSourceRhsJacobCell::execute() START\n");
  
  initializeComputationRHS();

  CFLog(VERBOSE, "After initializeComputationRHS\n");
  
  const CFuint nbCells = this->socket_states.getDataHandle().size();
  cf_assert(nbCells > 0);
  DataHandle<CFreal> updateCoeff = this->socket_updateCoeff.getDataHandle();
  DataHandle<CFreal> rhs = this->socket_rhs.getDataHandle();
  DataHandle<CFreal> normals = this->socket_normals.getDataHandle();
  DataHandle<CFint> isOutward = this->socket_isOutward.getDataHandle();  
  
  SafePtr<SCHEME> lf = this->getMethodData().getFluxSplitter().template d_castTo<SCHEME>();
  SafePtr<POLYREC> pr = this->getMethodData().getPolyReconstructor().template d_castTo<POLYREC>();
  SafePtr<LIMITER> lm = this->getMethodData().getLimiter().template d_castTo<LIMITER>();
  SafePtr<typename PHYSICS::PTERM> phys = PhysicalModelStack::getActive()->getImplementor()->
    getConvectiveTerm().d_castTo<typename PHYSICS::PTERM>();
  

  
  //Added for Source 
  
  SelfRegistPtr<SOURCE> ls1  = (*this->getMethodData().getSourceTermComputer())[0].template d_castTo<SOURCE>(); //Only valid if there is only one source term!!
  SafePtr<SOURCE> ls = ls1.getPtr();
  typedef typename SOURCE::template DeviceFunc<GPU, PHYSICS> SourceTerm; 



  typedef typename SCHEME::template  DeviceFunc<GPU, PHYSICS> FluxScheme;  
  typedef typename POLYREC::template DeviceFunc<PHYSICS> PolyRec;  
  typedef typename LIMITER::template DeviceFunc<PHYSICS> Limiter;  
  
  CudaEnv::CudaTimer& timer = CudaEnv::CudaTimer::getInstance();


  if (this->m_onGPU) {
    
    timer.start();
    // copy of data that change at every iteration
    this->socket_states.getDataHandle().getGlobalArray()->put(); 
    this->m_ghostStates.put();
    this->socket_volumes.getDataHandle().getLocalArray()->put(); 

    CFLog(VERBOSE, "FVMCC_ComputeSourceRhsJacobCell::execute() => CPU-->GPU data transfer took " << timer.elapsed() << " s\n");
    timer.start();
    
    ConfigOptionPtr<POLYREC, NOTYPE, GPU> dcor(pr);
    ConfigOptionPtr<LIMITER, NOTYPE, GPU> dcol(lm);
    ConfigOptionPtr<SCHEME,  NOTYPE, GPU> dcof(lf);
    ConfigOptionPtr<typename PHYSICS::PTERM, NOTYPE, GPU> dcop(phys);
    //Added for Source    
    ConfigOptionPtr<SOURCE, NOTYPE, GPU> dcos(ls);
    const CFuint blocksPerGrid = CudaEnv::CudaDeviceManager::getInstance().getBlocksPerGrid(nbCells);
    const CFuint nThreads = CudaEnv::CudaDeviceManager::getInstance().getNThreads();
    
    CFLog(VERBOSE, "FVMCC_ComputeSourceRhsJacobCell::execute() <blocksPerGrid, nThreads> = <" 
	  <<  blocksPerGrid << "," << nThreads << ">\n");
    
    //dim3 blocks(this->m_nbBlocksPerGridX, this->m_nbBlocksPerGridY);
    
    timer.start();
    
    // compute the cell-based gradients
    computeGradientsKernel<PHYSICS, PolyRec> <<<blocksPerGrid,nThreads>>> 
      (dcor.getPtr(),
       nbCells,
       this->socket_states.getDataHandle().getGlobalArray()->ptrDev(), 
       this->socket_nodes.getDataHandle().getGlobalArray()->ptrDev(),
       this->m_centerNodes.ptrDev(), 
       this->m_ghostStates.ptrDev(),
       this->m_ghostNodes.ptrDev(),
       this->socket_uX.getDataHandle().getLocalArray()->ptrDev(),
       this->socket_uY.getDataHandle().getLocalArray()->ptrDev(),
       this->socket_uZ.getDataHandle().getLocalArray()->ptrDev(),
       this->socket_limiter.getDataHandle().getLocalArray()->ptrDev(),
       updateCoeff.getLocalArray()->ptrDev(), 
       rhs.getLocalArray()->ptrDev(),
       normals.getLocalArray()->ptrDev(),
       isOutward.getLocalArray()->ptrDev(),
       this->m_cellInfo.ptrDev(),
       this->m_cellStencil.ptrDev(),
       this->m_cellFaces->getPtr()->ptrDev(),
       this->m_cellNodes->getPtr()->ptrDev(),
       this->m_neighborTypes.ptrDev(),
       this->m_cellConn.ptrDev());
    
    CFLog(VERBOSE, "FVMCC_ComputeSourceRhsJacobCell::execute() => computeGradientsKernel took " << timer.elapsed() << " s\n");
    
    timer.start();
    
    // compute the limiter in each cell
    computeLimiterKernel<PHYSICS, PolyRec, Limiter> <<<blocksPerGrid,nThreads>>> 
      (dcol.getPtr(), 
       dcor.getPtr(), 
       nbCells,
       this->socket_states.getDataHandle().getGlobalArray()->ptrDev(), 
       this->socket_nodes.getDataHandle().getGlobalArray()->ptrDev(),
       this->m_centerNodes.ptrDev(), 
       this->m_ghostStates.ptrDev(),
       this->m_ghostNodes.ptrDev(),
       this->socket_uX.getDataHandle().getLocalArray()->ptrDev(),
       this->socket_uY.getDataHandle().getLocalArray()->ptrDev(),
       this->socket_uZ.getDataHandle().getLocalArray()->ptrDev(),
       this->socket_limiter.getDataHandle().getLocalArray()->ptrDev(),
       updateCoeff.getLocalArray()->ptrDev(), 
       rhs.getLocalArray()->ptrDev(),
       normals.getLocalArray()->ptrDev(),
       isOutward.getLocalArray()->ptrDev(),
       this->m_cellInfo.ptrDev(),
       this->m_cellStencil.ptrDev(),
       this->m_cellFaces->getPtr()->ptrDev(),
       this->m_cellNodes->getPtr()->ptrDev(),
       this->m_neighborTypes.ptrDev(),
       this->m_cellConn.ptrDev());
    
    CFLog(VERBOSE, "FVMCC_ComputeSourceRhsJacobCell::execute() => computeLimiterKernel took " << timer.elapsed() << " s\n");
    
    timer.start();
    // compute the flux jacobian in each cell
    CFLog(VERBOSE, "FVMCC_ComputeSourceRhsJacobCell::execute() => Configuring method \n");
    ConfigOptionPtr<NumericalJacobian, PHYSICS, GPU> dcon
      (&this->getMethodData().getNumericalJacobian());
    CFreal startCellID = 0;
    CFreal FluxTime = 0;
    CFreal UpdateSystemTime = 0;
    

 CFLog(VERBOSE,"useParalutionPtr "<< this->m_useParalutionPtr << "\n");

if(!this->m_useParalutionPtr){


    CFLog(VERBOSE, "FVMCC_ComputeSourceRhsJacobCell::execute() => End of Configuring method \n");
    for (CFuint s = 0; s < m_nbCellsInKernel.size(); ++s) {
      CFLog(VERBOSE, "FVMCC_ComputeSourceRhsJacobCell::execute() => loop " << s << " of " << m_nbCellsInKernel.size() << "\n");

      CFreal ResFactor = this->getMethodData().getResFactor(); //Default = 1
      bool IsAxisymmetric = this->getMethodData().isAxisymmetric(); //Default = false
      computeFluxSourceJacobianKernel<FluxScheme, PolyRec, Limiter, SourceTerm> <<<m_nbKernelBlocks,nThreads>>> 
	(dcof.getPtr(),
	 dcor.getPtr(),
	 dcol.getPtr(),
	 dcon.getPtr(),
	 dcop.getPtr(),
         dcos.getPtr(),
         this->socket_volumes.getDataHandle().getLocalArray()->ptrDev(),
	 nbCells,
	 startCellID,
	 this->socket_states.getDataHandle().getGlobalArray()->ptrDev(), 
	 this->socket_nodes.getDataHandle().getGlobalArray()->ptrDev(),
	 this->m_centerNodes.ptrDev(), 
	 this->m_ghostStates.ptrDev(),
	 this->m_ghostNodes.ptrDev(),
	 m_blockJacobians.ptrDev(), 
	 m_blockStart.ptrDev(),
	 this->socket_uX.getDataHandle().getLocalArray()->ptrDev(),
	 this->socket_uY.getDataHandle().getLocalArray()->ptrDev(),
	 this->socket_uZ.getDataHandle().getLocalArray()->ptrDev(),
	 this->socket_limiter.getDataHandle().getLocalArray()->ptrDev(),
	 updateCoeff.getLocalArray()->ptrDev(), 
	 rhs.getLocalArray()->ptrDev(),
	 normals.getLocalArray()->ptrDev(),
	 isOutward.getLocalArray()->ptrDev(),
	 this->m_cellInfo.ptrDev(),
	 this->m_cellStencil.ptrDev(),
	 this->m_cellFaces->getPtr()->ptrDev(),
	 this->m_cellNodes->getPtr()->ptrDev(),
	 this->m_neighborTypes.ptrDev(),
	 this->m_cellConn.ptrDev(),
         ResFactor, 
         IsAxisymmetric);

      FluxTime += timer.elapsed();
  
      timer.start();
      m_blockJacobians.get();
      // update the portion of system matrix computed by this kernel
      updateSystemMatrix(s);
      startCellID += m_nbCellsInKernel[s];
      UpdateSystemTime += timer.elapsed();
      timer.start();
    }

    CFLog(VERBOSE, "FVMCC_ComputeSourceRhsJacobCell::execute() => computeFluxSourceJacobianKernel took " << FluxTime << "\n");
    CFLog(VERBOSE, "FVMCC_ComputeSourceRhsJacobCell::execute() => updateSystemMatrix took " << UpdateSystemTime << "\n");

}else{  /// New code for building the matrix on the GPU
/*  //  IA:  UNCOMMENT THIS FOR BUILDING THE SYSTEM ON THE GPU USING PARALUTION 2/2
#ifdef CF_HAVE_PARALUTION

 SafePtr<LSSMatrix> matrix = m_lss->getMatrix();
 SafePtr<Paralution::ParalutionMatrix> pmatrix = matrix.d_castTo<Paralution::ParalutionMatrix>();
// SafePtr<LSSVector> rhs;
// SafePtr<ParalutionVector> prhs = rhs.d_castTo<ParalutionVector>();


 //ParalutionVector rhs = m_lss->getRhs();  Need to implement this
 
 //CFreal* rowOffPtrDev = matrix->getRowOffPtrDev();
 //CFreal* colOffPtrDev = matrix->getColPtrDev();
 //CFreal* valPtrDev = matrix->getValPtrDev()
  
 //CFreal* rhsPtrDev = rhs->getPtrDev();


// ParalutionMatrix* matrix = m_lss->getMatrix();
  //Kernel Call
  CFreal ResFactor = 1.0;
  bool IsAxisymmetric = false;
for (CFuint s = 0; s < m_nbCellsInKernel.size(); ++s) {
computeFluxSourceJacobianKernelParalution<FluxScheme, PolyRec, Limiter, SourceTerm> <<<m_nbKernelBlocks,nThreads>>>
	(dcof.getPtr(),
	 dcor.getPtr(),
	 dcol.getPtr(),
	 dcon.getPtr(),
	 dcop.getPtr(),
         dcos.getPtr(),
         this->socket_volumes.getDataHandle().getLocalArray()->ptrDev(),
	 nbCells,
	 startCellID,
	 this->socket_states.getDataHandle().getGlobalArray()->ptrDev(), 
	 this->socket_nodes.getDataHandle().getGlobalArray()->ptrDev(),
	 this->m_centerNodes.ptrDev(), 
	 this->m_ghostStates.ptrDev(),
	 this->m_ghostNodes.ptrDev(),
//	 m_blockJacobians.ptrDev(), 
  	 m_blockStart.ptrDev(),
	 pmatrix->getRowoffPtrDev(),
	 pmatrix->getColPtrDev(),
	 pmatrix->getValPtrDev(),
	 this->socket_uX.getDataHandle().getLocalArray()->ptrDev(),
	 this->socket_uY.getDataHandle().getLocalArray()->ptrDev(),
	 this->socket_uZ.getDataHandle().getLocalArray()->ptrDev(),
	 this->socket_limiter.getDataHandle().getLocalArray()->ptrDev(),
	 updateCoeff.getLocalArray()->ptrDev(), 
	 rhs.getLocalArray()->ptrDev(),
	 normals.getLocalArray()->ptrDev(),
	 isOutward.getLocalArray()->ptrDev(),
	 this->m_cellInfo.ptrDev(),
	 this->m_cellStencil.ptrDev(),
	 this->m_cellFaces->getPtr()->ptrDev(),
	 this->m_cellNodes->getPtr()->ptrDev(),
	 this->m_neighborTypes.ptrDev(),
	 this->m_cellConn.ptrDev(),
         ResFactor, 
         IsAxisymmetric);
startCellID += m_nbCellsInKernel[s];
}
    m_blockJacobians.free();
    CFLog(VERBOSE, "FVMCC_ComputeSourceRhsJacobCell::execute() => computeFluxSourceJacobianKernelParalution took " << timer.elapsed() << "\n");
    timer.start();  

#endif
*/


}



    //m_blockJacobians.free();

    rhs.getLocalArray()->get();
    updateCoeff.getLocalArray()->get();
    CFLog(VERBOSE, "FVMCC_ComputeSourceRhsJacobCell::execute() => GPU-->CPU data transfer took " << timer.elapsed() << " s\n");
  }
  else {
    ConfigOptionPtr<SCHEME>  dcof(lf);
    ConfigOptionPtr<POLYREC> dcor(pr);
    ConfigOptionPtr<LIMITER> dcol(lm);
    ConfigOptionPtr<NumericalJacobian, PHYSICS> dcon(&this->getMethodData().getNumericalJacobian());
    ConfigOptionPtr<typename PHYSICS::PTERM> dcop(phys);
    ConfigOptionPtr<SOURCE> dcos(ls);
    
    CFreal ResFactor = this->getMethodData().getResFactor(); //Default = 1
    bool IsAxisymmetric = this->getMethodData().isAxisymmetric(); //Default = false
    
    computeFluxSourceJacobianCPU<FluxScheme, SourceTerm, PolyRec, Limiter>
      (dcof.getPtr(),
       dcor.getPtr(),
       dcol.getPtr(),
       dcon.getPtr(),
       dcop.getPtr(),
       dcos.getPtr(),
       nbCells,
       this->socket_states.getDataHandle().getGlobalArray()->ptr(), 
       this->socket_volumes.getDataHandle().getLocalArray()->ptr(),
       this->socket_nodes.getDataHandle().getGlobalArray()->ptr(),
       this->m_centerNodes.ptr(), 
       this->m_ghostStates.ptr(),
       this->m_ghostNodes.ptr(),
       m_blockJacobians.ptr(), 
       m_blockStart.ptr(),
       this->socket_uX.getDataHandle().getLocalArray()->ptr(),
       this->socket_uY.getDataHandle().getLocalArray()->ptr(),
       this->socket_uZ.getDataHandle().getLocalArray()->ptr(),
       this->socket_limiter.getDataHandle().getLocalArray()->ptr(),
       updateCoeff.getLocalArray()->ptr(), 
       rhs.getLocalArray()->ptr(),
       normals.getLocalArray()->ptr(),
       isOutward.getLocalArray()->ptr(),
       this->m_cellInfo.ptr(),
       this->m_cellStencil.ptr(),
       this->m_cellFaces->getPtr()->ptr(),
       this->m_cellNodes->getPtr()->ptr(),
       this->m_neighborTypes.ptr(),
       this->m_cellConn.ptr(),
       ResFactor, IsAxisymmetric);
    timer.start();
    // update the system matrix
    updateSystemMatrix(0);
    CFLog(VERBOSE, "FVMCC_ComputeSourceRhsJacobCell::execute() => updateSystemMatrix took " << timer.elapsed() << "\n");
  }


  timer.start();
  // compute flux jacobians on boundaries
  executeBC();
  CFLog(VERBOSE, "FVMCC_ComputeSourceRhsJacobCell::execute() => executeBC() took " << timer.elapsed() << " s\n");

  finalizeComputationRHS();
  
  CFLog(VERBOSE, "FVMCC_ComputeSourceRhsJacobCell::execute() END\n");

 /* // DEBUG
  for (int i = 0; i < rhs.size(); ++i) {
    std::cout << "updateCoeff[" << i << "] = " << updateCoeff[i]  << std::endl;
    std::cout << "rhs[" << i << "] = ";
    for (int j = 0; j < 18; ++j) {
     cout.precision(14); cout.setf(ios::scientific,ios::floatfield);  std::cout << rhs[i*18+j] << " ";
    }
    std::cout << std::endl;
  }
  
   abort();
 */ 
  CFTRACEEND;
}

//////////////////////////////////////////////////////////////////////////////

template <CFuint NBEQS>
void printGradients(CFreal* uX, CFreal* uY, CFreal* uZ, CFuint nbCells)
{  
  CFuint idxr = 0;
  for (CFuint cellID = 0; cellID < nbCells; ++cellID) {
    for (CFuint i = 0; i < NBEQS; ++i, ++idxr) {
      std::cout << "cellID["<< cellID << "], "<< i << " => UX (";
      std::cout.precision(12); std::cout << uX[idxr] << ", " << uY[idxr] << ", " << uZ[idxr] << ")\n";
    }
  } 
}

//////////////////////////////////////////////////////////////////////////////

template <CFuint NBEQS>
void printLimiter(CFreal* limiter, CFuint nbCells)
{ 
  CFuint idxl = 0;
  for (CFuint cellID = 0; cellID < nbCells; ++cellID) {
    std::cout << "cellID["<< cellID << "] => LIM (";
    for (CFuint i = 0; i < NBEQS; ++i, ++idxl) {
      std::cout.precision(12); std::cout << limiter[idxl] << " ";
    }
    std::cout << ")\n";
  }
}
 
//////////////////////////////////////////////////////////////////////////////

   } // namespace FiniteVolume
    
  } // namespace Numerics

} // namespace COOLFluiD

